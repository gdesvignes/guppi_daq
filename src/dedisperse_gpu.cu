#include "hip/hip_runtime.h"
/* dedisperse_gpu.cu 
 * Functions for coherent dedispersion in CUDA/GPUs
 * Paul Demorest, 2009/05
 */
#include "dedisperse_gpu.h"

/* CUDA kernel to convert bytes to floats.  Also splits incoming
 * data into two polarizations (assuming polns are interleaved
 * in the raw data).
 */
__global__ void byte_to_float_2pol_complex(
        unsigned short *in, float2 *outx, float2 *outy,
        size_t n) {
    const int nt = blockDim.x * gridDim.x;
    const int tId = blockIdx.x * blockDim.x + threadIdx.x;
    char4 *in_8bit = (char4 *)in;
    for (int i=tId; i<n; i+=nt) {
        outx[i].x = __int2float_rn(in_8bit[i].x);
        outx[i].y = __int2float_rn(in_8bit[i].y);
        outy[i].x = __int2float_rn(in_8bit[i].z);
        outy[i].y = __int2float_rn(in_8bit[i].w);
    }
}

/* CUDA kernel for inplace complex vector (elementwise) multiplication:
 *
 *   Does a[i] *= b[i] for 0<i<n.
 *
 *   a can contain multiple vectors to be multiplied by b, each
 *   should run in its own thread block, ie:
 *     vec_mult_complex<<<n_vector,64>>>(a,b,n_element_per_vector);
 *   where a has n_element_per_vector*n_vector entries, and b
 *   has n_element.
 */
#if 1 
__global__ void vector_multiply_complex(float2 *a, const float2 *b, 
        unsigned nelem) {
    const unsigned nelem_chunk = nelem / gridDim.y;
    const unsigned elem0 = blockIdx.y * nelem_chunk;
    const unsigned elem1 = elem0 + nelem_chunk > nelem ? 
        nelem : elem0 + nelem_chunk;
    float2 *ptr = &a[blockIdx.x*nelem];
    float2 tmp;
    for (int i=elem0+threadIdx.x; i<elem1; i+=blockDim.x) {
        tmp.x = ptr[i].x*b[i].x - ptr[i].y*b[i].y;
        tmp.y = ptr[i].y*b[i].x + ptr[i].x*b[i].y;
        ptr[i] = tmp;
    }
}
#endif
#if 0 
__global__ void vector_multiply_complex(float2 *a, const float2 *b, 
        unsigned nelem) {
    float2 *ptr = &a[blockIdx.x*nelem];
    float2 tmp;
    for (int i=threadIdx.x; i<nelem; i+=blockDim.x) {
        tmp.x = ptr[i].x*b[i].x - ptr[i].y*b[i].y;
        tmp.y = ptr[i].y*b[i].x + ptr[i].x*b[i].y;
        ptr[i] = tmp;
    }
}
#endif

/* Full-stokes detection "in-place" 
 * vx and vy arrays are voltage data.  Output total power
 * terms go into vx, and cross terms in vy.  n is total number
 * of data points.
 * TODO: check signs, etc
 * Also, if we're folding on the GPU it probably makes more sense
 * to combine the two operations.
 */
__global__ void detect_4pol(float2 *vx, float2 *vy, size_t n) {
    const int nt = blockDim.x * gridDim.x;
    const int tId = blockIdx.x * blockDim.x + threadIdx.x;
    float2 pp, xp;
    for (int i=tId; i<n; i+=nt) {
        pp.x = vx[i].x*vx[i].x + vx[i].y*vx[i].y;
        pp.y = vy[i].x*vy[i].x + vy[i].y*vy[i].y;
        xp.x = vx[i].x*vy[i].x + vx[i].y*vy[i].y;
        xp.y = vx[i].x*vy[i].y - vx[i].y*vy[i].x;
        vx[i] = pp;
        vy[i] = xp;
    }
}

/* Expand the original input out so that FFTs will overlap */
void expand_overlap(struct dedispersion_setup *s) {
    const size_t bytes_per_sample = 4; // 8-bit complex, 2 pol
    const size_t bytes_per_fft = bytes_per_sample * s->fft_len;
    const size_t bytes_overlap = bytes_per_sample * s->overlap;
    const size_t bytes_total = bytes_per_sample * s->npts_per_block;
    size_t icount=0, ocount=0;
    for (icount=0, ocount=0;
            icount<=bytes_total-bytes_per_fft; 
            icount+=bytes_per_fft-bytes_overlap,
            ocount+=bytes_per_fft)
        hipMemcpy(s->overlap_gpu + ocount, s->tbuf_tr_gpu + icount,
                bytes_per_fft, hipMemcpyDeviceToDevice);
}

/* Transfer the dedispersed data back to the main system memory
 * while simultaneously collapsing the overlap (ie, removing
 * invalid points at FFT edge).
 */
void transfer_collapse_overlap(struct dedispersion_setup *s) {
    /* At this point, databuf0 holds total-power terms (AA, BB)
     * and databuf1 holds poln cross-terms (Re, Im(AB)).
     */
    const int valid_samp_per_fft = s->fft_len - s->overlap;
    unsigned ifft;
    // TODO Think about the best way to organize this data for later
    // TODO Make sure we're getting the right part of the FFT
    for (ifft=0; ifft<s->nfft_per_block; ifft++) {
        // Each memcpy transfers a single FFT's worth of valid data
        // from 2 (out of 4 total) polns.  
        hipMemcpy(s->result_host + (4*ifft+0)*valid_samp_per_fft,
                s->databuf0_gpu + ifft*s->fft_len + s->overlap/2,
                valid_samp_per_fft * sizeof(float) * 2,
                hipMemcpyDeviceToHost); 
        hipMemcpy(s->result_host + (4*ifft+2)*valid_samp_per_fft,
                s->databuf1_gpu + ifft*s->fft_len + s->overlap/2,
                valid_samp_per_fft * sizeof(float) * 2,
                hipMemcpyDeviceToHost); 
    }

}


/* Coalesced transpose (or corner-turn) with no bank conflicts 
 * Assume two 8-bit complex polarizations
 * Each block transposes/copies a tile of tile_dim x tile_dim elements
 * using tile_dim x block_rows threads, so that each thread transposes
 * tile_dim/block_rows elements.  tile_dim must be an integral multiple of block_rows
 */
__global__ void corner_turn(char4 *odata, char4 *idata, int nchan, int npts_per_block, int tile_dim, int block_rows) {
    __shared__ char4 tile[16][17];

    int xIndex = blockIdx.x * tile_dim + threadIdx.x;
    int yIndex = blockIdx.y * tile_dim + threadIdx.y;
    int index_in = xIndex + (yIndex)*nchan;

    xIndex = blockIdx.y * tile_dim + threadIdx.x;
    yIndex = blockIdx.x * tile_dim + threadIdx.y;
    int index_out = xIndex + (yIndex)*npts_per_block;

    for (int i=0; i<tile_dim; i+=block_rows) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*nchan];
    }
    __syncthreads();

    for (int i=0; i<tile_dim; i+=block_rows) {
      odata[index_out+i*npts_per_block] = tile[threadIdx.x][threadIdx.y+i];
    }
}


/* Fills in the freq-domain chirp, given the input params.
 * Assumes memory has already been allocated.  If fft_len has not
 * been changed, this func can be called again to change the
 * DM, freq, etc currently being applied.  Formula is taken
 * from ASP's CalcChirp.
 */
void init_chirp(struct dedispersion_setup *s) {

    // Alloc temporary host memory
    float2 *chirp_host;
    size_t chirp_size = sizeof(float2) * s->fft_len * s->nchan;
    // TODO check that this isn't too big
    printf("sizeof(chirp_gpu) = %d MB\n", (int)(chirp_size >> 20));
    chirp_host = (float2 *)malloc(chirp_size);

    double dmfac = s->dm*2.0*M_PI/(2.41e-10*(1.0+s->earth_z4/1.0e4));
    double band_sign = (s->bw<0.0) ? -1.0 : 1.0;

    int i, ichan;
    for (ichan=0; ichan<s->nchan; ichan++) {

        for (i=0; i<s->fft_len; i++) {

            double dfreq = (double)i * s->bw / (double)s->fft_len;
            if (i>s->fft_len/2) dfreq -= s->bw;

            double freqfac = dfreq / s->freq[ichan];
            freqfac = freqfac * freqfac / (s->freq[ichan] + dfreq);

            double arg = band_sign * dmfac * freqfac;

            // This makes Ingrid happy, but I have no idea where this
            // particular formula comes from.
            // double taper = 1.0/sqrt(1.0 + pow(fabs(dfreq)/(0.47*s->bw),80));
            double taper = 1.0;

            chirp_host[ichan*s->fft_len + i].x = 
                (float)(cos(arg)*taper/(double)s->fft_len);
            chirp_host[ichan*s->fft_len + i].y = 
                -1.0*(float)(sin(arg)*taper/(double)s->fft_len);
#if 0
            chirp_host[ichan*s->fft_len + i].x = 1.0/(double)s->fft_len;
            chirp_host[ichan*s->fft_len + i].y = 0.0;
#endif
        }

    }

    // Transfer the values to the gpu, free host memory
    hipMemcpy(s->chirp_gpu[0], chirp_host, chirp_size, hipMemcpyHostToDevice);
    free(chirp_host);
}

/* Initialize all necessary memory, etc for doing dedispersion 
 * at the given params.  In the struct, the following MUST be 
 * filled in:
 *   rf, bw, dm, npts_per_block, npol
 * Optionally, fft_len and overlap can be specified as well.  If
 * either of these is set to 0, it will be computed automatically
 * from the input params.
 * TODO: more error checking
 */
extern "C"
void init_dedispersion(struct dedispersion_setup *s) {

    // Find lowest freq
    int i;
    double f_chan_lo_mhz = s->freq[0];
    for (i=1; i<s->nchan; i++) 
        if (s->freq[i] < f_chan_lo_mhz) 
            f_chan_lo_mhz = s->freq[i];

    printf("rf=%f bw=%f dm=%f freq_lo=%f\n", s->rf, s->bw, s->dm,
            f_chan_lo_mhz);

    // Calc various parameters
    double f_lo_ghz = (f_chan_lo_mhz - fabs(s->bw)/2.0)/1.0e3;
    double f_hi_ghz = (f_chan_lo_mhz + fabs(s->bw)/2.0)/1.0e3;
    double chirp_len_samples = 4150. * s->dm *
        (1.0/(f_lo_ghz*f_lo_ghz) - 1.0/(f_hi_ghz*f_hi_ghz));
    printf("Chirp length = %f us\n", chirp_len_samples);
    chirp_len_samples *= fabs(s->bw);
    printf("Chirp length = %f samples\n", chirp_len_samples);

    if (s->overlap==0 && chirp_len_samples!=0.0) {
        // Do nearest power of 2 for now.  Find out what's optimal
        // Also find out what values don't work.
        s->overlap=1;
        while (s->overlap<chirp_len_samples) s->overlap <<= 1;
    }

    if (s->fft_len==0) {
        // Rough optimization based on testing w/ CUDA 2.3
        // Could make a "dedispersion plan" that tests?
        s->fft_len = 16*1024;
        if (s->overlap <= 1024) s->fft_len = 32*1024; // previously 16
        else if (s->overlap <= 2048) s->fft_len = 64*1024;
        else if (s->overlap <= 16*1024) s->fft_len = 128*1024;
        else if (s->overlap <= 64*1024) s->fft_len = 256*1024;
        while (s->fft_len < 2.0*s->overlap) s->fft_len *= 2;
        if (s->fft_len > 8*1024*1024) {
            printf("init_dedispersion error: FFT length too large! (%d)\n",
                    s->fft_len);
            s->fft_len = 8*1024*1024;
        }

    }

    printf("fft_len=%d overlap=%d\n", s->fft_len, s->overlap); fflush(stdout);

    // Figure out the transpose parameters
    if (s->nchan==4) {
        s->tile_dim = 4;
        s->block_rows = 4;
    } else if (s->nchan==8) {
        s->tile_dim = 8;
        s->block_rows = 8;
    } else if (s->nchan>=16) {
        s->tile_dim = 16;
        s->block_rows = 8;
    } else {	
        printf("Nchan (%d) not supported!\n", s->nchan);
    }	

    // Figure out how many FFTs per block
    s->nfft_per_block = 1;
    int npts_used = s->fft_len;
    while(npts_used <= s->npts_per_block) {
        s->nfft_per_block++;
        npts_used = s->nfft_per_block*(s->fft_len-s->overlap) + s->overlap;
    }
    s->nfft_per_block--;
    npts_used = s->nfft_per_block*(s->fft_len-s->overlap) + s->overlap;

    // Allocate memory
    const size_t bytes_per_sample = 4; // 8-bit complex 2-pol
    const size_t bytes_in = bytes_per_sample * s->npts_per_block;
    const size_t bytes_tot = bytes_per_sample * s->fft_len*s->nfft_per_block;
    const size_t bytes_databuf = sizeof(float2)*s->fft_len*s->nfft_per_block;
    const size_t bytes_chirp = sizeof(float2)*s->fft_len*s->nchan;
    size_t total_gpu_mem = 0;

    printf("npts_per_block=%d nfft_per_block=%d npts_used=%d diff=%d\n",
            s->npts_per_block, s->nfft_per_block, npts_used,
            s->npts_per_block - npts_used); 
    fflush(stdout);

    hipError_t rv = hipHostAlloc((void**)&(s->tbuf_host), bytes_in, 
            hipHostMallocWriteCombined);
    hipMalloc((void**)&s->tbuf_gpu, bytes_in);
    hipMalloc((void**)&s->tbuf_tr_gpu, bytes_in);
    total_gpu_mem += 2*bytes_in;
    hipMalloc((void**)&s->overlap_gpu, bytes_tot);
    total_gpu_mem += bytes_tot;
    hipMalloc((void**)&s->databuf0_gpu, 2 * bytes_databuf);
    s->databuf1_gpu = s->databuf0_gpu + s->fft_len*s->nfft_per_block;
    total_gpu_mem += 2*bytes_databuf;
    hipMalloc((void**)&s->chirp_gpu[0], bytes_chirp);
    total_gpu_mem += bytes_chirp;
    for (i=0; i<s->nchan; i++) s->chirp_gpu[i] = s->chirp_gpu[0] + i*s->fft_len;

    printf("allocated mem\n"); fflush(stdout);
    printf("total_gpu_mem = %d MB\n", total_gpu_mem >> 20);

    hipDeviceSynchronize();
    printf("init_dedispersion2 cuda_err=\'%s\'\n", hipGetErrorString(hipGetLastError()));
    // Init chirp function
    init_chirp(s);
    printf("chirp \n"); fflush(stdout);

    hipDeviceSynchronize();
    printf("init_dedispersion3 cuda_err=\'%s\'\n", hipGetErrorString(hipGetLastError()));

    // Plan FFT
    // nfft_per_block is only for 1 pol, hence the factor of 2 here
    hipfftResult fft_rv = 
        hipfftPlan1d(&s->plan, s->fft_len, HIPFFT_C2C, 2*s->nfft_per_block);
    printf("fft (%d)\n", fft_rv); fflush(stdout);

    hipDeviceSynchronize();
    printf("init_dedispersion4 cuda_err=\'%s\'\n", hipGetErrorString(hipGetLastError()));

    // Zero out fold buffers (call init_fold() to set them up)
    s->fold_phase = NULL;
    s->fold_step = NULL;
    s->foldbuf_gpu = NULL;

    // Zero out ds buffer
    s->dsbuf_gpu = NULL;

    // Init timers
    init_timers(&s->time);

    // Check errors
    hipDeviceSynchronize();
    printf("init_dedispersion cuda_err=\'%s\'\n", hipGetErrorString(hipGetLastError()));

}

/* Init the set of timers */
void init_timers(struct dedispersion_times *t) {
    int it;
    memset(t,0,sizeof(struct dedispersion_times));
    for (it=0; it<NTIMERS; it++) hipEventCreate(&(t->t[it]));
}

/* Free the set of timers */
void free_timers(struct dedispersion_times *t) {
    int it;
    for (it=0; it<NTIMERS; it++) hipEventDestroy(t->t[it]); 
}

/* Accumulate times into appropriate places */
#define get_time(idx0,idx1,step) do { \
    hipEventElapsedTime(&ttmp, t->t[idx0], t->t[idx1]); \
    rv = hipGetLastError(); \
    if (rv==hipSuccess) t->step += ttmp; \
} while (0)
void accumulate_timers(struct dedispersion_times *t) {
    // Note, needs to be consistent with markers in processing fns
   
    float ttmp; 
    hipError_t rv;

    // Wait for the final event, it's either 10 or 16 depending on mode
    hipEventSynchronize(t->t[10]);
    hipEventSynchronize(t->t[16]);

    get_time(0,1,transfer_to_gpu);
    get_time(1,2,corner_turn);
    get_time(2,3,overlap);
    get_time(3,4,bit_to_float);
    get_time(4,5,fft);
    get_time(5,6,xmult);
    get_time(6,7,fft);
    get_time(8,9,downsample);
    get_time(9,10,transfer_to_host);
    get_time(12,13,fold_mem);
    get_time(13,14,fold_blocks);
    get_time(14,15,fold_combine);
    get_time(15,16,transfer_to_host);

    // Only one of these two will succeed
    get_time(0,10,total);
    get_time(0,16,total);
}

/* Actually do the dedispersion */
extern "C"
void dedisperse(struct dedispersion_setup *s, int ichan,
        const unsigned char *in, float *out) {


    hipfftResult fft_rv;

    /* Various sizes */
    const size_t bytes_per_sample = 4; // 8-bit complex 2-pol
    const size_t bytes_in = bytes_per_sample * s->npts_per_block;
    const size_t npts_tot = s->fft_len*s->nfft_per_block;

    /* copy input data to transfer buffer */
    memcpy(s->tbuf_host, in, bytes_in);

    hipEventRecord(s->time.t[0]); // Start

    /* Copy data to GPU */
    hipMemcpy(s->tbuf_gpu, s->tbuf_host, bytes_in, hipMemcpyHostToDevice);
    hipEventRecord(s->time.t[1]); // Finish HtoD

    /* Corner turn the data */
    dim3 grid(s->nchan/s->tile_dim, s->npts_per_block/s->tile_dim), threads(s->tile_dim, s->block_rows);
    corner_turn<<<grid, threads>>>((char4 *)s->tbuf_tr_gpu, (char4 *)s->tbuf_gpu, s->nchan, s->npts_per_block, s->tile_dim, s->block_rows);
    hipEventRecord(s->time.t[2]); // Finish Corner turn

    /* Expand overlap */
    expand_overlap(s);
    hipEventRecord(s->time.t[3]); // Finish overlap

    /* Convert to floating point */
    byte_to_float_2pol_complex<<<16,128>>>((unsigned short *)s->overlap_gpu, 
            s->databuf0_gpu, s->databuf1_gpu, npts_tot);
    hipEventRecord(s->time.t[4]); // Finish covert

    /* Forward FFT */
    fft_rv = hipfftExecC2C(s->plan, s->databuf0_gpu, s->databuf0_gpu, 
            HIPFFT_FORWARD);
    hipEventRecord(s->time.t[5]); // Finish FFT

    /* Multiply by chirp */
    dim3 gd(2*s->nfft_per_block, s->fft_len/4096, 1);
    //dim3 gd(2*s->nfft_per_block, 1, 1);
    vector_multiply_complex<<<gd,64>>>(s->databuf0_gpu,
            s->chirp_gpu[ichan], s->fft_len);
    hipEventRecord(s->time.t[6]); // Finish xmult

    /* Inverse FFT */
    fft_rv = hipfftExecC2C(s->plan, s->databuf0_gpu, s->databuf0_gpu, 
            HIPFFT_BACKWARD);
    hipEventRecord(s->time.t[7]); // Finish IFFT

    int nvalid = s->nfft_per_block*(s->fft_len-s->overlap);
    s->time.nsamp_tot += nvalid;

}

/* Free any resources associated with dedispersion */
extern "C"
void free_dedispersion(struct dedispersion_setup *s) {

    hipDeviceSynchronize(); // Need?

    free_timers(&s->time);

    hipHostFree(s->tbuf_host);
    hipFree(s->tbuf_gpu);
    hipFree(s->overlap_gpu);
    hipFree(s->databuf0_gpu);
    hipFree(s->chirp_gpu[0]);

    hipDeviceReset();
}

#define print_percent(var) \
    printf("  %.3f ns %7.2f%% %s\n", \
            1e6*s->time.var/(double)s->time.nsamp_tot, \
            100.0*s->time.var/s->time.total, #var)
#define print_percent_short(var) \
    fprintf(f, "%.3f ", s->time.var/s->time.total)
void print_timing_report(struct dedispersion_setup *s) {

    /* Print to screen */
    printf("\n");
    printf("Total time  = %6.1f s (%.4f ns/samp)\n", 
            s->time.total/1e3, 1e6*s->time.total/(double)s->time.nsamp_tot);
    printf("Total2 time = %6.1f s (%.4f ns/samp)\n", 
            s->time.total2/1e3, 1e6*s->time.total2/(double)s->time.nsamp_tot);
    print_percent(transfer_to_gpu);
    print_percent(corner_turn);
    print_percent(overlap);
    print_percent(bit_to_float);
    print_percent(fft);
    print_percent(xmult);
    print_percent(fold_mem);
    print_percent(fold_blocks);
    print_percent(fold_combine);
    print_percent(downsample);
    print_percent(transfer_to_host);

#if 0 
    /* print short version to file */
    FILE *f = fopen("dedisp_timing.dat", "a");
    fprintf(f, "%7d %6d %.4e %.4e ",  s->fft_len, s->overlap,
            s->time.total/(double)s->time.nsamp_tot,
            s->gp->drop_frac_tot);
    print_percent_short(transfer_to_gpu);
    print_percent_short(corner_turn);
    print_percent_short(overlap);
    print_percent_short(bit_to_float);
    print_percent_short(fft);
    print_percent_short(xmult);
    print_percent_short(fold_mem);
    print_percent_short(fold_blocks);
    print_percent_short(fold_combine);
    print_percent_short(downsample);
    print_percent_short(transfer_to_host);
    fprintf(f, "\n");
    fclose(f);
#endif
}
